#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>

#include <nv_metrics.h>

#define CHECK_CUDA_ERRORS(call)                                                \
  {                                                                            \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      std::cerr << "CUDA error in " << #call << " function (" << __FILE__      \
                << ":" << __LINE__ << "): " << hipGetErrorString(error)       \
                << std::endl;                                                  \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

__global__ void kernel(float *a, float *b, float *c, float n, float scale) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = c[i] + scale * (a[i] + b[i]);
  }
}

int main() {
  const int N = 1024;
  float *a, *b, *c;
  float *d_a, *d_b, *d_c;
  const float scale = 2.0f;

  // Allocate memory on host
  a = new float[N];
  b = new float[N];
  c = new float[N];

  // Allocate memory on device
  CHECK_CUDA_ERRORS(hipMalloc((void **)&d_a, N * sizeof(float)));
  CHECK_CUDA_ERRORS(hipMalloc((void **)&d_b, N * sizeof(float)));
  CHECK_CUDA_ERRORS(hipMalloc((void **)&d_c, N * sizeof(float)));

  // Initialize host vectors
  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  // Copy host vectors to device
  CHECK_CUDA_ERRORS(
      hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERRORS(
      hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice));

  // Setup metrics
  std::vector<std::string> metrics = {
      "sm__sass_thread_inst_executed_op_fadd_pred_on.sum",
      "sm__sass_thread_inst_executed_op_fmul_pred_on.sum",
      "sm__sass_thread_inst_executed_op_ffma_pred_on.sum"};

  // Launch configuration
  const int block_size = 256;
  const int grid_size = (N + block_size - 1) / block_size;

  // Start measurement
  nvmetrics::measureMetricsStart(metrics);

  // Launch kernel
  kernel<<<grid_size, block_size>>>(d_a, d_b, d_c, N, scale);

  // Stop measurement
  std::vector<double> result = nvmetrics::measureMetricsStop();
  assert(metrics.size() == result.size());

  // Check for kernel launch errors
  CHECK_CUDA_ERRORS(hipGetLastError());

  // Print result of the measurement
  for (int i = 0; i < result.size(); i++) {
    std::cout << metrics[i] << ": " << result[i] << std::endl;
  }

  // Copy result back to host
  CHECK_CUDA_ERRORS(
      hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

  // Free device memory
  CHECK_CUDA_ERRORS(hipFree(d_a));
  CHECK_CUDA_ERRORS(hipFree(d_b));
  CHECK_CUDA_ERRORS(hipFree(d_c));

  // Free host memory
  delete[] a;
  delete[] b;
  delete[] c;

  return EXIT_SUCCESS;
}
